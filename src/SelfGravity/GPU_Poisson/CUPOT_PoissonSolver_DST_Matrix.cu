#include "hip/hip_runtime.h"
#include "Macro.h"
#include "CUPOT.h"

#if ( defined GRAVITY  &&  defined GPU  &&  POT_SCHEME == DST )


#define POT_NXT_F    ( PATCH_SIZE+2*POT_GHOST_SIZE           )
#define POT_PAD      ( WARP_SIZE/2 - (POT_NXT_F*2%WARP_SIZE) )
#define POT_NTHREAD  ( RHO_NXT*RHO_NXT/2    )
#define POT_USELESS  ( POT_GHOST_SIZE%2                      )

#if (RHO_NXT==24)
  __device__ real M[24*24] = {0.12533323356430426 , 0.2486898871648548 , 0.3681245526846779 , 0.4817536741017153 , 0.5877852522924731 , 0.6845471059286886 , 0.7705132427757891 , 0.8443279255020151 , 0.9048270524660196 , 0.9510565162951535 , 0.9822872507286886 , 0.9980267284282716 , 0.9980267284282716 , 0.9822872507286887 , 0.9510565162951536 , 0.9048270524660195 , 0.844327925502015 , 0.7705132427757893 , 0.6845471059286888 , 0.5877852522924732 , 0.4817536741017152 , 0.36812455268467814 , 0.24868988716485524 , 0.12533323356430454 , 0.2486898871648548 , 0.4817536741017153 , 0.6845471059286886 , 0.8443279255020151 , 0.9510565162951535 , 0.9980267284282716 , 0.9822872507286887 , 0.9048270524660195 , 0.7705132427757893 , 0.5877852522924732 , 0.36812455268467814 , 0.12533323356430454 , -0.1253332335643043 , -0.3681245526846779 , -0.5877852522924727 , -0.7705132427757894 , -0.9048270524660198 , -0.9822872507286887 , -0.9980267284282716 , -0.9510565162951536 , -0.8443279255020151 , -0.684547105928689 , -0.4817536741017161 , -0.24868988716485535 , 0.3681245526846779 , 0.6845471059286886 , 0.9048270524660196 , 0.9980267284282716 , 0.9510565162951536 , 0.7705132427757893 , 0.4817536741017152 , 0.12533323356430454 , -0.24868988716485457 , -0.5877852522924727 , -0.8443279255020147 , -0.9822872507286887 , -0.9822872507286886 , -0.8443279255020151 , -0.587785252292474 , -0.24868988716485535 , 0.12533323356430506 , 0.48175367410171493 , 0.7705132427757887 , 0.9510565162951532 , 0.9980267284282716 , 0.90482705246602 , 0.6845471059286884 , 0.368124552684678 , 0.4817536741017153 , 0.8443279255020151 , 0.9980267284282716 , 0.9048270524660195 , 0.5877852522924732 , 0.12533323356430454 , -0.3681245526846779 , -0.7705132427757894 , -0.9822872507286887 , -0.9510565162951536 , -0.684547105928689 , -0.24868988716485535 , 0.2486898871648549 , 0.6845471059286886 , 0.9510565162951532 , 0.9822872507286886 , 0.7705132427757886 , 0.368124552684678 , -0.12533323356430318 , -0.5877852522924728 , -0.9048270524660197 , -0.9980267284282716 , -0.8443279255020161 , -0.48175367410171627 , 0.5877852522924731 , 0.9510565162951535 , 0.9510565162951536 , 0.5877852522924732 , 1.2246467991473532e-16 , -0.5877852522924727 , -0.9510565162951535 , -0.9510565162951536 , -0.5877852522924732 , -2.4492935982947064e-16 , 0.5877852522924722 , 0.9510565162951532 , 0.9510565162951536 , 0.5877852522924734 , 3.6739403974420594e-16 , -0.5877852522924728 , -0.9510565162951534 , -0.9510565162951538 , -0.5877852522924735 , -4.898587196589413e-16 , 0.5877852522924728 , 0.9510565162951529 , 0.9510565162951543 , 0.5877852522924751 , 0.6845471059286886 , 0.9980267284282716 , 0.7705132427757893 , 0.12533323356430454 , -0.5877852522924727 , -0.9822872507286887 , -0.8443279255020151 , -0.24868988716485535 , 0.48175367410171493 , 0.9510565162951532 , 0.90482705246602 , 0.368124552684678 , -0.3681245526846789 , -0.9048270524660197 , -0.9510565162951543 , -0.48175367410171627 , 0.24868988716485635 , 0.8443279255020146 , 0.9822872507286889 , 0.5877852522924751 , -0.1253332335643047 , -0.7705132427757879 , -0.9980267284282716 , -0.6845471059286887 , 0.7705132427757891 , 0.9822872507286887 , 0.4817536741017152 , -0.3681245526846779 , -0.9510565162951535 , -0.8443279255020151 , -0.12533323356430467 , 0.6845471059286886 , 0.9980267284282716 , 0.5877852522924734 , -0.24868988716485302 , -0.9048270524660197 , -0.9048270524660185 , -0.2486898871648556 , 0.5877852522924714 , 0.9980267284282716 , 0.6845471059286886 , -0.1253332335643047 , -0.8443279255020135 , -0.9510565162951538 , -0.36812455268467664 , 0.4817536741017121 , 0.9822872507286884 , 0.7705132427757888 , 0.8443279255020151 , 0.9048270524660195 , 0.12533323356430454 , -0.7705132427757894 , -0.9510565162951536 , -0.24868988716485535 , 0.6845471059286886 , 0.9822872507286886 , 0.368124552684678 , -0.5877852522924728 , -0.9980267284282716 , -0.48175367410171627 , 0.4817536741017155 , 0.9980267284282716 , 0.5877852522924751 , -0.3681245526846787 , -0.982287250728689 , -0.6845471059286887 , 0.2486898871648527 , 0.9510565162951533 , 0.7705132427757888 , -0.12533323356430268 , -0.9048270524660179 , -0.8443279255020163 , 0.9048270524660196 , 0.7705132427757893 , -0.24868988716485457 , -0.9822872507286887 , -0.5877852522924732 , 0.48175367410171493 , 0.9980267284282716 , 0.368124552684678 , -0.684547105928688 , -0.9510565162951538 , -0.12533323356430578 , 0.8443279255020146 , 0.8443279255020152 , -0.1253332335643047 , -0.9510565162951534 , -0.6845471059286887 , 0.3681245526846786 , 0.9980267284282714 , 0.48175367410171666 , -0.5877852522924724 , -0.9822872507286887 , -0.24868988716485782 , 0.7705132427757877 , 0.9048270524660202 , 0.9510565162951535 , 0.5877852522924732 , -0.5877852522924727 , -0.9510565162951536 , -2.4492935982947064e-16 , 0.9510565162951532 , 0.5877852522924734 , -0.5877852522924728 , -0.9510565162951538 , -4.898587196589413e-16 , 0.9510565162951529 , 0.5877852522924751 , -0.5877852522924726 , -0.9510565162951538 , -7.347880794884119e-16 , 0.9510565162951533 , 0.5877852522924738 , -0.5877852522924724 , -0.9510565162951539 , -9.797174393178826e-16 , 0.9510565162951532 , 0.5877852522924769 , -0.5877852522924694 , -0.951056516295155 , 0.9822872507286886 , 0.36812455268467814 , -0.8443279255020153 , -0.684547105928689 , 0.5877852522924729 , 0.9048270524660192 , -0.24868988716485474 , -0.9980267284282716 , -0.12533323356430404 , 0.9510565162951534 , 0.4817536741017165 , -0.7705132427757901 , -0.7705132427757888 , 0.4817536741017152 , 0.951056516295155 , -0.12533323356430268 , -0.9980267284282717 , -0.24868988716485438 , 0.9048270524660179 , 0.587785252292474 , -0.68454710592869 , -0.8443279255020164 , 0.3681245526846748 , 0.9822872507286882 , 0.9980267284282716 , 0.12533323356430454 , -0.9822872507286887 , -0.24868988716485535 , 0.9510565162951532 , 0.368124552684678 , -0.9048270524660197 , -0.48175367410171627 , 0.8443279255020146 , 0.5877852522924751 , -0.7705132427757879 , -0.6845471059286887 , 0.6845471059286902 , 0.7705132427757888 , -0.5877852522924695 , -0.8443279255020163 , 0.4817536741017181 , 0.9048270524660202 , -0.3681245526846749 , -0.951056516295155 , 0.24868988716485566 , 0.9822872507286895 , -0.1253332335643057 , -0.9980267284282716 , 0.9980267284282716 , -0.1253332335643043 , -0.9822872507286887 , 0.2486898871648549 , 0.9510565162951536 , -0.36812455268467725 , -0.90482705246602 , 0.4817536741017155 , 0.8443279255020152 , -0.5877852522924726 , -0.770513242775791 , 0.6845471059286876 , 0.6845471059286887 , -0.7705132427757878 , -0.5877852522924739 , 0.8443279255020153 , 0.48175367410171377 , -0.9048270524660194 , -0.3681245526846804 , 0.9510565162951532 , 0.24868988716485124 , -0.9822872507286876 , -0.125333233564305 , 0.9980267284282713 , 0.9822872507286887 , -0.3681245526846779 , -0.8443279255020151 , 0.6845471059286886 , 0.5877852522924734 , -0.9048270524660197 , -0.2486898871648556 , 0.9980267284282716 , -0.1253332335643047 , -0.9510565162951538 , 0.4817536741017121 , 0.7705132427757888 , -0.7705132427757924 , -0.48175367410171677 , 0.9510565162951522 , 0.12533323356430465 , -0.9980267284282716 , 0.24868988716485566 , 0.9048270524660219 , -0.587785252292472 , -0.6845471059286866 , 0.8443279255020112 , 0.36812455268468075 , -0.9822872507286889 , 0.9510565162951536 , -0.5877852522924727 , -0.5877852522924732 , 0.9510565162951532 , 3.6739403974420594e-16 , -0.9510565162951538 , 0.5877852522924728 , 0.5877852522924751 , -0.9510565162951534 , -7.347880794884119e-16 , 0.951056516295155 , -0.5877852522924724 , -0.587785252292471 , 0.9510565162951532 , 1.102182119232618e-15 , -0.951056516295155 , 0.587785252292475 , 0.5877852522924741 , -0.9510565162951521 , -1.4695761589768238e-15 , 0.951056516295153 , -0.5877852522924661 , -0.5877852522924774 , 0.951056516295153 , 0.9048270524660195 , -0.7705132427757894 , -0.24868988716485535 , 0.9822872507286886 , -0.5877852522924728 , -0.48175367410171627 , 0.9980267284282716 , -0.3681245526846787 , -0.6845471059286887 , 0.9510565162951533 , -0.12533323356430268 , -0.8443279255020163 , 0.8443279255020153 , 0.12533323356430465 , -0.951056516295155 , 0.6845471059286898 , 0.3681245526846739 , -0.9980267284282716 , 0.4817536741017115 , 0.5877852522924744 , -0.9822872507286889 , 0.24868988716485174 , 0.770513242775794 , -0.9048270524660176 , 0.844327925502015 , -0.9048270524660198 , 0.12533323356430418 , 0.7705132427757886 , -0.9510565162951534 , 0.24868988716485463 , 0.68454710592869 , -0.982287250728689 , 0.3681245526846786 , 0.5877852522924738 , -0.9980267284282714 , 0.481753674101715 , 0.48175367410171377 , -0.9980267284282718 , 0.5877852522924693 , 0.3681245526846739 , -0.9822872507286888 , 0.6845471059286896 , 0.2486898871648584 , -0.9510565162951541 , 0.770513242775794 , 0.12533323356430887 , -0.9048270524660207 , 0.8443279255020147 , 0.7705132427757893 , -0.9822872507286887 , 0.48175367410171493 , 0.368124552684678 , -0.9510565162951538 , 0.8443279255020146 , -0.1253332335643047 , -0.6845471059286887 , 0.9980267284282714 , -0.5877852522924724 , -0.24868988716485782 , 0.9048270524660202 , -0.9048270524660194 , 0.24868988716485566 , 0.5877852522924741 , -0.9980267284282716 , 0.6845471059286896 , 0.12533323356430864 , -0.8443279255020167 , 0.951056516295153 , -0.3681245526846776 , -0.48175367410172076 , 0.9822872507286896 , -0.770513242775787 , 0.6845471059286888 , -0.9980267284282716 , 0.7705132427757887 , -0.12533323356430318 , -0.5877852522924735 , 0.9822872507286889 , -0.8443279255020155 , 0.2486898871648527 , 0.48175367410171666 , -0.9510565162951539 , 0.9048270524660179 , -0.3681245526846749 , -0.3681245526846771 , 0.9048270524660189 , -0.9510565162951521 , 0.4817536741017115 , 0.24868988716485152 , -0.8443279255020167 , 0.9822872507286882 , -0.5877852522924716 , -0.12533323356430548 , 0.7705132427757941 , -0.9980267284282716 , 0.684547105928684 , 0.5877852522924732 , -0.9510565162951536 , 0.9510565162951532 , -0.5877852522924728 , -4.898587196589413e-16 , 0.5877852522924751 , -0.9510565162951538 , 0.9510565162951533 , -0.5877852522924724 , -9.797174393178826e-16 , 0.5877852522924769 , -0.951056516295155 , 0.9510565162951532 , -0.587785252292472 , -1.4695761589768238e-15 , 0.5877852522924744 , -0.9510565162951541 , 0.951056516295153 , -0.5877852522924716 , -1.959434878635765e-15 , 0.5877852522924748 , -0.9510565162951564 , 0.9510565162951506 , -0.5877852522924655 , 0.4817536741017152 , -0.8443279255020151 , 0.9980267284282716 , -0.9048270524660197 , 0.5877852522924728 , -0.1253332335643047 , -0.36812455268467664 , 0.7705132427757888 , -0.9822872507286887 , 0.9510565162951532 , -0.6845471059286847 , 0.24868988716485566 , 0.24868988716485124 , -0.6845471059286866 , 0.9510565162951552 , -0.9822872507286889 , 0.770513242775794 , -0.3681245526846776 , -0.12533323356430548 , 0.5877852522924748 , -0.9048270524660177 , 0.9980267284282709 , -0.8443279255020126 , 0.4817536741017169 , 0.36812455268467814 , -0.684547105928689 , 0.9048270524660192 , -0.9980267284282716 , 0.9510565162951534 , -0.7705132427757901 , 0.4817536741017152 , -0.12533323356430268 , -0.24868988716485438 , 0.587785252292474 , -0.8443279255020164 , 0.9822872507286882 , -0.9822872507286889 , 0.8443279255020151 , -0.5877852522924661 , 0.24868988716485174 , 0.12533323356430182 , -0.48175367410171455 , 0.7705132427757941 , -0.9510565162951543 , 0.9980267284282718 , -0.9048270524660174 , 0.6845471059286837 , -0.36812455268468347 , 0.24868988716485524 , -0.4817536741017161 , 0.6845471059286884 , -0.8443279255020161 , 0.9510565162951532 , -0.9980267284282716 , 0.9822872507286884 , -0.9048270524660179 , 0.7705132427757877 , -0.5877852522924751 , 0.3681245526846748 , -0.1253332335643057 , -0.12533323356429796 , 0.36812455268468075 , -0.5877852522924774 , 0.770513242775794 , -0.9048270524660207 , 0.9822872507286896 , -0.9980267284282716 , 0.9510565162951551 , -0.8443279255020126 , 0.6845471059286837 , -0.48175367410171366 , 0.24868988716485765 , 0.12533323356430454 , -0.24868988716485535 , 0.368124552684678 , -0.48175367410171627 , 0.5877852522924751 , -0.6845471059286887 , 0.7705132427757888 , -0.8443279255020163 , 0.9048270524660202 , -0.951056516295155 , 0.9822872507286895 , -0.9980267284282716 , 0.9980267284282718 , -0.9822872507286889 , 0.9510565162951509 , -0.9048270524660176 , 0.8443279255020186 , -0.770513242775787 , 0.684547105928684 , -0.5877852522924655 , 0.4817536741017169 , -0.36812455268467026 , 0.24868988716485765 , -0.12533323356430429};

#elif (RHO_NXT==16)
  __device__ real M[16*16]= {0.18374951781657034 , 0.3612416661871529 , 0.5264321628773557 , 0.6736956436465572 , 0.7980172272802395 , 0.8951632913550623 , 0.961825643172819 , 0.9957341762950345 , 0.9957341762950346 , 0.961825643172819 , 0.8951632913550626 ,0.7980172272802396 , 0.6736956436465571 , 0.5264321628773561 , 0.3612416661871533 , 0.18374951781657037 , 0.3612416661871529 , 0.6736956436465572 , 0.8951632913550623 , 0.9957341762950345 , 0.961825643172819 , 0.7980172272802396 , 0.5264321628773561 , 0.18374951781657037 , -0.18374951781657015 , -0.5264321628773558 , -0.7980172272802388 , -0.961825643172819 , -0.9957341762950345 , -0.8951632913550626 , -0.6736956436465578 , -0.361241666187153 , 0.5264321628773557 , 0.8951632913550623 , 0.9957341762950346 , 0.7980172272802396 , 0.3612416661871533 , -0.18374951781657015 , -0.6736956436465572 , -0.961825643172819 , -0.961825643172819 , -0.6736956436465578 , -0.18374951781657092 , 0.3612416661871526 , 0.7980172272802399 , 0.9957341762950345, 0.8951632913550635 , 0.5264321628773563 , 0.6736956436465572 , 0.9957341762950345 , 0.7980172272802396 , 0.18374951781657037 , -0.5264321628773558 , -0.961825643172819 , -0.8951632913550626 , -0.361241666187153 , 0.3612416661871526 , 0.8951632913550623 , 0.9618256431728196 , 0.5264321628773563 , -0.1837495178165712 , -0.7980172272802387 , -0.9957341762950347 , -0.6736956436465573 , 0.7980172272802395 , 0.961825643172819 , 0.3612416661871533 , -0.5264321628773558 , -0.9957341762950346 , -0.6736956436465578 , 0.18374951781656956 , 0.8951632913550623 , 0.8951632913550627 , 0.18374951781657017 , -0.6736956436465568 , -0.9957341762950347 , -0.5264321628773548 , 0.3612416661871515 , 0.9618256431728189 , 0.7980172272802394 , 0.8951632913550623 , 0.7980172272802396 , -0.18374951781657015 , -0.961825643172819 , -0.6736956436465578 , 0.3612416661871526 , 0.9957341762950345 , 0.5264321628773563 , -0.5264321628773557 , -0.9957341762950347 , -0.3612416661871541 , 0.6736956436465567 , 0.9618256431728187 , 0.18374951781657042 , -0.7980172272802365 , -0.8951632913550628 , 0.961825643172819 , 0.5264321628773561 , -0.6736956436465572 , -0.8951632913550626 , 0.18374951781656956 , 0.9957341762950345 , 0.361241666187154 , -0.7980172272802387 ,-0.7980172272802393 , 0.3612416661871515 , 0.9957341762950347 , 0.18374951781657042 , -0.8951632913550638 , -0.6736956436465589 , 0.5264321628773523 , 0.9618256431728197 , 0.9957341762950345 , 0.18374951781657037 , -0.961825643172819 , -0.361241666187153 , 0.8951632913550623 , 0.5264321628773563 , -0.7980172272802387 , -0.6736956436465573 , 0.6736956436465567 , 0.7980172272802394 , -0.5264321628773524 , -0.8951632913550628 , 0.3612416661871546 , 0.9618256431728197 , -0.18374951781656723 , -0.9957341762950346 , 0.9957341762950346 , -0.18374951781657015 , -0.961825643172819 , 0.3612416661871526 , 0.8951632913550627 , -0.5264321628773557 , -0.7980172272802393 , 0.6736956436465567 , 0.6736956436465588 , -0.7980172272802386 , -0.5264321628773566 , 0.8951632913550622 , 0.36124166618715275 , -0.9618256431728193 , -0.1837495178165725 , 0.9957341762950344 , 0.961825643172819 , -0.5264321628773558 , -0.6736956436465578 , 0.8951632913550623 , 0.18374951781657017 , -0.9957341762950347 , 0.3612416661871515 , 0.7980172272802394 , -0.7980172272802386 , -0.36124166618715264 , 0.9957341762950344 , -0.18374951781656723 , -0.8951632913550613 , 0.6736956436465549 , 0.526432162877357 , -0.9618256431728192 , 0.8951632913550626 , -0.7980172272802388 , -0.18374951781657006 , 0.9618256431728196 , -0.673695643646558 , -0.3612416661871524 , 0.9957341762950345 , -0.5264321628773524 , -0.5264321628773566 , 0.9957341762950347 , -0.3612416661871512 , -0.6736956436465564 , 0.9618256431728203 , -0.18374951781657048 , -0.7980172272802418 , 0.8951632913550588 , 0.7980172272802396 , -0.961825643172819 , 0.3612416661871526 , 0.5264321628773563 , -0.9957341762950347 , 0.6736956436465567 , 0.18374951781657042 , -0.8951632913550628 , 0.8951632913550622, -0.18374951781656723 , -0.673695643646559 , 0.9957341762950344 , -0.5264321628773581 , -0.3612416661871531 , 0.9618256431728218 , -0.7980172272802382 , 0.6736956436465571 , -0.9957341762950345 , 0.7980172272802394 , -0.1837495178165712 , -0.5264321628773564 , 0.9618256431728192 , -0.8951632913550622 , 0.3612416661871546 , 0.36124166618715275 , -0.8951632913550629 , 0.9618256431728183 , -0.5264321628773551 , -0.18374951781656926 , 0.7980172272802398 , -0.9957341762950344 , 0.6736956436465599, 0.5264321628773561 , -0.8951632913550626 , 0.9957341762950345 , -0.7980172272802387 , 0.3612416661871515 , 0.18374951781657042 , -0.6736956436465589 , 0.9618256431728197 , -0.9618256431728193 , 0.6736956436465549 , -0.18374951781656698 , -0.3612416661871531 , 0.7980172272802355 , -0.9957341762950349 , 0.8951632913550587 , -0.5264321628773516 , 0.3612416661871533 , -0.6736956436465578 , 0.8951632913550627 , -0.9957341762950347 , 0.9618256431728189 , -0.7980172272802386 , 0.5264321628773553 ,-0.18374951781656723 , -0.1837495178165725 , 0.526432162877357 , -0.7980172272802418 , 0.9618256431728199 , -0.995734176295035 , 0.8951632913550618 , -0.6736956436465571 , 0.36124166618714704 , 0.18374951781657037 , -0.361241666187153 , 0.5264321628773563 , -0.6736956436465573 , 0.7980172272802394 , -0.8951632913550628 , 0.9618256431728197 , -0.9957341762950346 , 0.9957341762950344 , -0.9618256431728192 , 0.8951632913550588 , -0.7980172272802382 , 0.6736956436465599 , -0.5264321628773516 , 0.36124166618714704 , -0.18374951781656976};

#endif


/************************************************************
  Many optimization options for SOR are defined in CUPOT.h
************************************************************/


// variables reside in constant memory
#include "CUDA_ConstMemory.h"

extern __shared__  unsigned char shared_mem[];

__device__ uint Rhoid_3Dto1D(uint x, uint y, uint z, uint N, uint XYZ){

  if (XYZ==0) return __umul24(z , N*N) + __umul24(y , N) + x;
  if (XYZ==1) return __umul24(z , N*N) + __umul24(x , N) + y;
  if (XYZ==2) return __umul24(x , N*N) + __umul24(z , N) + y;

  return 0;
}

__device__ void DST_Scheme(const uint ID,
			   uint Nslab,
			   real *Rho_Array,
			   typename FFT_DST::workspace_type workspace,
			   uint XYZ){

  uint t,Rhoid_x,Rhoid_y,Rhoid_z,Rhoid,Rhoid_r;
  uint N2 = cufftdx::size_of<FFT_DST>::value;
  uint N = (cufftdx::size_of<FFT_DST>::value / 2) - 1 ;
  uint NC = (cufftdx::size_of<FFT_DST>::value / 2) + 1 ;
  uint Nstride = N/Nslab;
  uint stride = blockDim.x * blockDim.y;
  real c;


  for (int step=0; step<RHO_NXT/2; step++){

    t = ID ;
    stride = blockDim.x * blockDim.y;
    do 
    {

     Rhoid_z = t / (N*N);
     Rhoid_y = t / N % N;
     Rhoid_x = t % N ;

     //     if (Rhoid_z < 1){
     Rhoid =  Rhoid_3Dto1D( Rhoid_x, 
			    Rhoid_y, 
			    Rhoid_z + step *2,
			    N,XYZ);
       
     reinterpret_cast<real_type*>(shared_mem)[t ]  =   Rho_Array[Rhoid];
     t += stride;
     
    } while (t < N*N*N/RHO_NXT*2);
    __syncthreads();
     
    t=ID;
    do 
    {

     Rhoid_z = t / (N*N);
     Rhoid_y = t / N % N;
     Rhoid_x = t % N ;
     Rhoid =  Rhoid_3Dto1D( Rhoid_x, 
			    Rhoid_y, 
			    Rhoid_z + step *2,
			    N,XYZ);

     c = 0;
     for (int e=0; e<N; e++)
       c += reinterpret_cast<real_type*>(shared_mem)[(Rhoid_z * N + Rhoid_y)*N + e] * M[Rhoid_x * N + e];
     
     Rho_Array[Rhoid] = c * 2.0;
     t += stride;

    } while (t < N*N*N/RHO_NXT*2);
    __syncthreads();

  } //   for (int step=0; step<RHO_NXT; step++){

    __syncthreads();
}

__device__ void Assign_sFPot(   real TempFPot,int FID,
				int bid,
				int FIDxx, int FIDyy,int FIDzz)

{
  
    int  FIDz = FID /(POT_NXT_F*POT_NXT_F);
    int  FIDy = (FID /POT_NXT_F) % POT_NXT_F ;
    int  FIDx = FID % POT_NXT_F;
    
    if (FIDx==0         and FIDy >= 1 and FIDy <= RHO_NXT and FIDz >= 1 and FIDz <= RHO_NXT) reinterpret_cast<real_type*>(shared_mem)[RHO_NXT*RHO_NXT * 0 + (FIDz-1)*RHO_NXT + FIDy-1 ] = TempFPot; 
    if (FIDy==0         and FIDx >= 1 and FIDx <= RHO_NXT and FIDz >= 1 and FIDz <= RHO_NXT) reinterpret_cast<real_type*>(shared_mem)[RHO_NXT*RHO_NXT * 1 + (FIDz-1)*RHO_NXT + FIDx-1] = TempFPot;
    if (FIDz==0         and FIDx >= 1 and FIDx <= RHO_NXT and FIDy >= 1 and FIDy <= RHO_NXT) reinterpret_cast<real_type*>(shared_mem)[RHO_NXT*RHO_NXT * 2 + (FIDy-1)*RHO_NXT + FIDx-1] = TempFPot;
      
    if (FIDx==RHO_NXT+1 and FIDy >= 1 and FIDy <= RHO_NXT and FIDz >= 1 and FIDz <= RHO_NXT) reinterpret_cast<real_type*>(shared_mem)[RHO_NXT*RHO_NXT * 3 + (FIDz-1)*RHO_NXT + FIDy-1] = TempFPot;
    if (FIDy==RHO_NXT+1 and FIDx >= 1 and FIDx <= RHO_NXT and FIDz >= 1 and FIDz <= RHO_NXT) reinterpret_cast<real_type*>(shared_mem)[RHO_NXT*RHO_NXT * 4 + (FIDz-1)*RHO_NXT + FIDx-1] = TempFPot;
    if (FIDz==RHO_NXT+1 and FIDx >= 1 and FIDx <= RHO_NXT and FIDy >= 1 and FIDy <= RHO_NXT) reinterpret_cast<real_type*>(shared_mem)[RHO_NXT*RHO_NXT * 5 + (FIDy-1)*RHO_NXT + FIDx-1] = TempFPot;

    //    if (FIDx==0 and FIDy-1==0 and FIDz-1==0 and bid==0) printf("Pot=%f %d %d %d %d\n",TempFPot,FID,FIDxx,FIDyy,FIDzz);

}
//-------------------------------------------------------------------------------------------------------
// Function    :  CUPOT_PoissonSolver_SOR
// Description :  GPU Poisson solver using the SOR scheme
//
// Note        :  1. Take advantage of shared memory
//                2. Prefix "g" for pointers pointing to the "Global" memory space
//                   Prefix "s" for pointers pointing to the "Shared" memory space
//                3. Each patch requires about 3.1*10^6 FLOPS (including the gravity solver)
//                   --> 133 GFLOPS is achieved in one C2050 GPU
//                4. Reference: Numerical Recipes, Chapter 20.5
//                5. Chester Cheng has implemented the SOR_USE_SHUFFLE and SOR_USE_PADDING optimizations, which
//                   greatly improve performance for PATCH_SIZE=8 && POT_GHOST_SIZE=5
//                6. Typically, the number of iterations required to reach round-off errors is 20 ~ 25 (single precision)
//                   for PATCH_SIZE=8 && POT_GHOST_SIZE=5
//
// Padding     :  Below shows how bank conflict is eliminated by padding.
//
//                Example constants :
//                      POT_NXT_F = 18                       // The number of floating point elements per row
//                      POT_PAD   = 16 - (18 * 2 % 32) = 12  // number of floating point elements that needs to be added
//                                                           // within thread groups
//
//                We now show how shared memory (s_FPot array) is accessed by a warp in residual evaluation.
//
//                Before Padding:
//                Thread number   |  Accessed shared memory bank
//                      00 ~ 07   |    | 01 |    | 03 |    | 05 |    | 07 |    | 09 |    | 11 |    | 13 |    | 15 |    |    |
//                      08 ~ 15   |    |    | 02 |    | 04 |    | 06 |    | 08 |    | 10 |    | 12 |    | 14 |    | 16 |    |
//                      16 ~ 23   |    | 05 |    | 07 |    | 09 |    | 11 |    | 13 |    | 15 |    | 17 |    | 19 |    |    |
//                      24 ~ 31   |    |    | 06 |    | 08 |    | 10 |    | 12 |    | 14 |    | 16 |    | 18 |    | 20 |    |
//
//                After Padding:
//                Thread number   |  Accessed shared memory bank
//                      00 ~ 07   |    | 01 |    | 03 |    | 05 |    | 07 |    | 09 |    | 11 |    | 13 |    | 15 |    |    |
//                      08 ~ 15   |    |    | 02 |    | 04 |    | 06 |    | 08 |    | 10 |    | 12 |    | 14 |    | 16 |    |
//                                ----------------- PAD 12 FLOATING POINTS HERE !!!!! ---------------------------------------
//                      16 ~ 23   |    | 17 |    | 19 |    | 21 |    | 23 |    | 25 |    | 27 |    | 29 |    | 31 |    |    |
//                      24 ~ 31   |    |    | 18 |    | 20 |    | 22 |    | 24 |    | 26 |    | 28 |    | 30 |    | 00 |    |
//
//
//                Additional Notes for Padding:
//                      1. When threads 08 ~ 15 access the elements below them (+y direction), we have to skip the padded
//                         elements. Same for when threads 16~23 access the elements above them (-y direction).
//                      2. For every warp we need to pad #PAD_POT floating point elements. Each xy plane has 4 warps working
//                         on it, so for each xy plane we need to pad #4*PAD_POT floating point elements.
//
//                 
// Parameter   :  g_Rho_Array     : Global memory array to store the input density
//                g_Pot_Array_In  : Global memory array storing the input "coarse-grid" potential for
//                                  interpolation
//                g_Pot_Array_Out : Global memory array to store the output potential
//                Min_Iter        : Minimum # of iterations for SOR
//                Max_Iter        : Maximum # of iterations for SOR
//                Omega_6         : Omega / 6
//                Const           : (Coefficient in front of the RHS in the Poisson eq.) / dh^2
//                IntScheme       : Interpolation scheme for potential
//                                  --> currently supported schemes include
//                                      INT_CQUAD : conservative quadratic interpolation
//                                      INT_QUAD  : quadratic interpolation
//---------------------------------------------------------------------------------------------------
//                           
__global__ void CUPOT_PoissonSolver_DST(       real g_Rho_Array    [][ RHO_NXT*RHO_NXT*RHO_NXT ], // RHO_NXT = 16 24 
                                               real g_Pot_Array_In [][ POT_NXT*POT_NXT*POT_NXT ], // POT_NXT = 12 16
                                               real g_Pot_Array_Out[][ GRA_NXT*GRA_NXT*GRA_NXT ], // GRA_NXT = 12 20
                                         const real Const, 
					 const IntScheme_t IntScheme,
					 typename FFT_DST::workspace_type workspace)
{
  //    printf("gridDim.x=%d gridDim.y=%d\n",gridDim.x ,gridDim.y);
  //  printf("%d %d %d %d\n",RHO_NXT, POT_NXT, GRA_NXT, POT_NXT_F);

  // const uint       input_size       = FFT::ffts_per_block * Nslab * cufftdx::size_of<FFT>::value;
  // real_type input_data[ Maxbid ][ input_size ];
  // __syncthreads();

  // const uint          output_size       = FFT::ffts_per_block * Nslab * (cufftdx::size_of<FFT>::value / 2 + 1);
  // complex_type output_data[ Maxbid ][ output_size ];
  // //  printf("output_size=%d\n",output_size);

  //   extern __shared__  unsigned char shared_mem[];


   const uint bid       = blockIdx.x;
   const uint tid_x     = threadIdx.x;
   const uint tid_y     = threadIdx.y;
   const uint tid_z     = threadIdx.z;
   const uint bdim_x    = blockDim.x;
   const uint bdim_y    = blockDim.y;
   const uint bdim_z    = blockDim.z;
   const uint ID        = __umul24( tid_z, __umul24(bdim_x,bdim_y) ) + __umul24( tid_y, bdim_x ) + tid_x;
   const uint dx        = 1;
   const uint dy        = POT_NXT_F;
   const uint dz        = POT_NXT_F*POT_NXT_F;
   const uint DispEven  = ( tid_y + tid_z ) & 1;
   const uint DispOdd   = DispEven^1;
   const uint DispFlip  = bdim_z & 1;
   const uint RhoID0    = __umul24( tid_z, RHO_NXT*RHO_NXT ) + __umul24( tid_y, RHO_NXT )+ ( tid_x << 1 );
   const uint dRhoID    = __umul24( bdim_z, RHO_NXT*RHO_NXT );
#  ifdef SOR_USE_PADDING
   const uint dPotID    = __umul24( bdim_z, POT_NXT_F*POT_NXT_F + POT_PAD*4 );
   const uint warpID    = ID % WARP_SIZE;
   const uint pad_dy_0  = ( warpID >=  8 && warpID <= 15 ) ? dy + POT_PAD : dy;    //
   const uint pad_dy_1  = ( warpID >= 16 && warpID <= 23 ) ? dy + POT_PAD : dy;    // please refer to the Padding notes above!
   const uint pad_dz    = dz + POT_PAD*4;                                          //
   const uint pad_pot   = ( tid_y < 2 ) ? 0 : POT_PAD*((tid_y-2)/4 + 1);
#  else
   const uint dPotID    = __umul24( bdim_z, POT_NXT_F*POT_NXT_F );
   const uint pad_dy_0  = dy;
   const uint pad_dy_1  = dy;
   const uint pad_dz    = dz;
   const uint pad_pot   = 0;
#  endif
   const uint PotID0    = pad_pot + __umul24( 1+tid_z, pad_dz ) + __umul24( 1+tid_y, dy ) + ( tid_x << 1 ) + 1;

   uint t, s_index;
   uint s_id,s_idr, stride;
   uint Rhoid, Rhoid_x,Rhoid_y,Rhoid_z;

// #  ifdef SOR_CPOT_SHARED
//    __shared__ real s_CPot[ POT_NXT  *POT_NXT  *POT_NXT   ];
//    printf("have s_CPot ??\n");
//    printf("POT_NXT=%d\n"POT_NXT);
// #  endif


// a1. load the fine-grid density into the shared memory
// -----------------------------------------------------------------------------------------------------------


// a2. load the coarse-grid potential into the shared memory
// -----------------------------------------------------------------------------------------------------------
#  ifdef SOR_CPOT_SHARED
   t = ID;
   do {  s_CPot[t] = g_Pot_Array_In[bid][t];    t += POT_NTHREAD; }     while ( t < POT_NXT*POT_NXT*POT_NXT );
   __syncthreads();
#  else
   const real *s_CPot = g_Pot_Array_In[bid];
#  endif

// b. evaluate the "fine-grid" potential by interpolation (as the initial guess and the B.C.)
// -----------------------------------------------------------------------------------------------------------
   const int N_CSlice = POT_NTHREAD / ( (POT_NXT-2)*(POT_NXT-2) );

   if ( ID < N_CSlice*(POT_NXT-2)*(POT_NXT-2) )
   {
      const real Const_8   = 1.0/8.0;
      const real Const_64  = 1.0/64.0;
      const real Const_512 = 1.0/512.0;

      const int Cdx  = 1;
      const int Cdy  = POT_NXT;
      const int Cdz  = POT_NXT*POT_NXT;
      const int CIDx = 1 + ID % ( POT_NXT-2 );
      const int CIDy = 1 + (  ID % ( (POT_NXT-2)*(POT_NXT-2) )  ) / ( POT_NXT-2 );
      const int CIDz = 1 + ID / ( (POT_NXT-2)*(POT_NXT-2) );
      int       CID  = __mul24( CIDz, Cdz ) + __mul24( CIDy, Cdy ) + __mul24( CIDx, Cdx );
      const int Fdx  = 1;
      const int Fdy  = POT_NXT_F;
      const int FIDx = ( (CIDx-1)<<1 ) - POT_USELESS;
      const int FIDy = ( (CIDy-1)<<1 ) - POT_USELESS;
      int       FIDz = ( (CIDz-1)<<1 ) - POT_USELESS;
#     ifdef SOR_USE_PADDING
      const int Fpad = ( FIDy < 3 ) ? 0 : POT_PAD*((FIDy-3)/4 + 1);    // padding logic
      const int Fdz  = POT_NXT_F*POT_NXT_F + POT_PAD*4;                // added padding
#     else
      const int Fpad = 0;
      const int Fdz  = POT_NXT_F*POT_NXT_F;
#     endif
      int       FID  = Fpad + __mul24( FIDz, Fdz ) + __mul24( FIDy, Fdy ) + __mul24( FIDx, Fdx );

      real TempFPot1, TempFPot2, TempFPot3, TempFPot4, TempFPot5, TempFPot6, TempFPot7, TempFPot8;
      real Slope_00, Slope_01, Slope_02, Slope_03, Slope_04, Slope_05, Slope_06, Slope_07;
      real Slope_08, Slope_09, Slope_10, Slope_11, Slope_12;
      int  Idx, Idy, Idz, ii, jj, kk;


      for (int z=CIDz; z<POT_NXT-1; z+=N_CSlice)
      {
         switch ( IntScheme )
         {
            /*
            case INT_CENTRAL :
            {
               Slope_00 = (real)0.125 * ( s_CPot[CID+Cdx] - s_CPot[CID-Cdx] );
               Slope_01 = (real)0.125 * ( s_CPot[CID+Cdy] - s_CPot[CID-Cdy] );
               Slope_02 = (real)0.125 * ( s_CPot[CID+Cdz] - s_CPot[CID-Cdz] );

               TempFPot1 = s_CPot[CID] - Slope_00 - Slope_01 - Slope_02;
               TempFPot2 = s_CPot[CID] + Slope_00 - Slope_01 - Slope_02;
               TempFPot3 = s_CPot[CID] - Slope_00 + Slope_01 - Slope_02;
               TempFPot4 = s_CPot[CID] + Slope_00 + Slope_01 - Slope_02;
               TempFPot5 = s_CPot[CID] - Slope_00 - Slope_01 + Slope_02;
               TempFPot6 = s_CPot[CID] + Slope_00 - Slope_01 + Slope_02;
               TempFPot7 = s_CPot[CID] - Slope_00 + Slope_01 + Slope_02;
               TempFPot8 = s_CPot[CID] + Slope_00 + Slope_01 + Slope_02;
            }
            break; // INT_CENTRAL
            */


            case INT_CQUAD :
            {
               Slope_00 = Const_8   * ( s_CPot[CID+Cdx        ] - s_CPot[CID-Cdx        ] );
               Slope_01 = Const_8   * ( s_CPot[CID    +Cdy    ] - s_CPot[CID    -Cdy    ] );
               Slope_02 = Const_8   * ( s_CPot[CID        +Cdz] - s_CPot[CID        -Cdz] );

               Slope_03 = Const_64  * ( s_CPot[CID+Cdx    -Cdz] - s_CPot[CID-Cdx    -Cdz] );
               Slope_04 = Const_64  * ( s_CPot[CID    +Cdy-Cdz] - s_CPot[CID    -Cdy-Cdz] );
               Slope_05 = Const_64  * ( s_CPot[CID+Cdx-Cdy    ] - s_CPot[CID-Cdx-Cdy    ] );
               Slope_06 = Const_64  * ( s_CPot[CID+Cdx+Cdy    ] - s_CPot[CID-Cdx+Cdy    ] );
               Slope_07 = Const_64  * ( s_CPot[CID+Cdx    +Cdz] - s_CPot[CID-Cdx    +Cdz] );
               Slope_08 = Const_64  * ( s_CPot[CID    +Cdy+Cdz] - s_CPot[CID    -Cdy+Cdz] );

               Slope_09 = Const_512 * ( s_CPot[CID+Cdx-Cdy-Cdz] - s_CPot[CID-Cdx-Cdy-Cdz] );
               Slope_10 = Const_512 * ( s_CPot[CID+Cdx+Cdy-Cdz] - s_CPot[CID-Cdx+Cdy-Cdz] );
               Slope_11 = Const_512 * ( s_CPot[CID+Cdx-Cdy+Cdz] - s_CPot[CID-Cdx-Cdy+Cdz] );
               Slope_12 = Const_512 * ( s_CPot[CID+Cdx+Cdy+Cdz] - s_CPot[CID-Cdx+Cdy+Cdz] );

               TempFPot1 = - Slope_00 - Slope_01 - Slope_02 - Slope_03 - Slope_04 - Slope_05 + Slope_06
                           + Slope_07 + Slope_08 - Slope_09 + Slope_10 + Slope_11 - Slope_12 + s_CPot[CID];

               TempFPot2 = + Slope_00 - Slope_01 - Slope_02 + Slope_03 - Slope_04 + Slope_05 - Slope_06
                           - Slope_07 + Slope_08 + Slope_09 - Slope_10 - Slope_11 + Slope_12 + s_CPot[CID];

               TempFPot3 = - Slope_00 + Slope_01 - Slope_02 - Slope_03 + Slope_04 + Slope_05 - Slope_06
                           + Slope_07 - Slope_08 + Slope_09 - Slope_10 - Slope_11 + Slope_12 + s_CPot[CID];

               TempFPot4 = + Slope_00 + Slope_01 - Slope_02 + Slope_03 + Slope_04 - Slope_05 + Slope_06
                           - Slope_07 - Slope_08 - Slope_09 + Slope_10 + Slope_11 - Slope_12 + s_CPot[CID];

               TempFPot5 = - Slope_00 - Slope_01 + Slope_02 + Slope_03 + Slope_04 - Slope_05 + Slope_06
                           - Slope_07 - Slope_08 + Slope_09 - Slope_10 - Slope_11 + Slope_12 + s_CPot[CID];

               TempFPot6 = + Slope_00 - Slope_01 + Slope_02 - Slope_03 + Slope_04 + Slope_05 - Slope_06
                           + Slope_07 - Slope_08 - Slope_09 + Slope_10 + Slope_11 - Slope_12 + s_CPot[CID];

               TempFPot7 = - Slope_00 + Slope_01 + Slope_02 + Slope_03 - Slope_04 + Slope_05 - Slope_06
                           - Slope_07 + Slope_08 - Slope_09 + Slope_10 + Slope_11 - Slope_12 + s_CPot[CID];

               TempFPot8 = + Slope_00 + Slope_01 + Slope_02 - Slope_03 - Slope_04 - Slope_05 + Slope_06
                           + Slope_07 + Slope_08 + Slope_09 - Slope_10 - Slope_11 + Slope_12 + s_CPot[CID];
            }
            break; // INT_CQUAD

            case INT_QUAD :
            {
               TempFPot1 = TempFPot2 = TempFPot3 = TempFPot4 = (real)0.0;
               TempFPot5 = TempFPot6 = TempFPot7 = TempFPot8 = (real)0.0;

               for (int dk=-1; dk<=1; dk++)  {  Idz = dk+1;    kk = __mul24( dk, Cdz );
               for (int dj=-1; dj<=1; dj++)  {  Idy = dj+1;    jj = __mul24( dj, Cdy );
               for (int di=-1; di<=1; di++)  {  Idx = di+1;    ii = __mul24( di, Cdx );

                  TempFPot1 += s_CPot[CID+kk+jj+ii] * c_Mm[Idz] * c_Mm[Idy] * c_Mm[Idx];
                  TempFPot2 += s_CPot[CID+kk+jj+ii] * c_Mm[Idz] * c_Mm[Idy] * c_Mp[Idx];
                  TempFPot3 += s_CPot[CID+kk+jj+ii] * c_Mm[Idz] * c_Mp[Idy] * c_Mm[Idx];
                  TempFPot4 += s_CPot[CID+kk+jj+ii] * c_Mm[Idz] * c_Mp[Idy] * c_Mp[Idx];
                  TempFPot5 += s_CPot[CID+kk+jj+ii] * c_Mp[Idz] * c_Mm[Idy] * c_Mm[Idx];
                  TempFPot6 += s_CPot[CID+kk+jj+ii] * c_Mp[Idz] * c_Mm[Idy] * c_Mp[Idx];
                  TempFPot7 += s_CPot[CID+kk+jj+ii] * c_Mp[Idz] * c_Mp[Idy] * c_Mm[Idx];
                  TempFPot8 += s_CPot[CID+kk+jj+ii] * c_Mp[Idz] * c_Mp[Idy] * c_Mp[Idx];

               }}}
            }
            break; // INT_QUAD

         } // switch ( IntScheme )

//       save data to the shared-memory array.
//       Currently this part is highly diverged. However, since the interpolation takes much less time than the
//       SOR iteration does, we have not yet tried to optimize this part

         if ( FIDz >= 0 )
         {
            if ( FIDx >= 0            &&  FIDy >= 0           )   Assign_sFPot(TempFPot1, FID,bid,FIDx,FIDy,FIDz);;
            if ( FIDx <= POT_NXT_F-2  &&  FIDy >= 0           )   Assign_sFPot(TempFPot2, FID+Fdx,bid,FIDx,FIDy,FIDz);
            if ( FIDx >= 0            &&  FIDy <= POT_NXT_F-2 )   Assign_sFPot(TempFPot3, FID    +Fdy,bid,FIDx,FIDy,FIDz);
            if ( FIDx <= POT_NXT_F-2  &&  FIDy <= POT_NXT_F-2 )   Assign_sFPot(TempFPot4, FID+Fdx+Fdy  ,bid,FIDx,FIDy,FIDz);
         }
	 
         if ( FIDz <= POT_NXT_F-2 )
         {
            if ( FIDx >= 0            &&  FIDy >= 0           )   Assign_sFPot(TempFPot5, FID        +Fdz,bid,FIDx,FIDy,FIDz);
            if ( FIDx <= POT_NXT_F-2  &&  FIDy >= 0           )   Assign_sFPot(TempFPot6, FID+Fdx    +Fdz,bid,FIDx,FIDy,FIDz);
            if ( FIDx >= 0            &&  FIDy <= POT_NXT_F-2 )   Assign_sFPot(TempFPot7, FID    +Fdy+Fdz,bid,FIDx,FIDy,FIDz);
            if ( FIDx <= POT_NXT_F-2  &&  FIDy <= POT_NXT_F-2 )   Assign_sFPot(TempFPot8, FID+Fdx+Fdy+Fdz,bid,FIDx,FIDy,FIDz);
         }


	 
         CID  += __mul24(   N_CSlice, Cdz );
         FID  += __mul24( 2*N_CSlice, Fdz );
         FIDz += 2*N_CSlice;

      } // for (int z=CIDz; z<POT_NXT-1; z+=N_CSlice)
   } // if ( ID < N_CSlice*(POT_NXT-2)*(POT_NXT-2) )
   __syncthreads();
   
   float bc_xm,bc_xp,bc_ym,bc_yp,bc_zm,bc_zp;
   unsigned int N = (cufftdx::size_of<FFT_DST>::value / 2) - 1 ;
         
// allocation shared_mem = [0 a b c d e f g 0 -g -f -e -d -c -b -a ] for DST from rho = [a b c d e f g]
   real temp;
   uint Nstride = RHO_NXT/Nslab;

   t = ID;
   stride = blockDim.x * blockDim.y;

   do 
   {
     //     if (t < RHO_NXT*RHO_NXT*RHO_NXT/Nslab * (stepB+1)){
     if (t < RHO_NXT*RHO_NXT*RHO_NXT){
     Rhoid_x = t % RHO_NXT;
     Rhoid_y = t/RHO_NXT % RHO_NXT;
     Rhoid_z = t / (RHO_NXT*RHO_NXT);

     s_id = __umul24(Rhoid_z, N*N)
          + __umul24(Rhoid_y, N)
          + Rhoid_x;
	     
     // if boundary condition
     if (Rhoid_x==0)         {bc_xm = reinterpret_cast<real_type*>(shared_mem)[RHO_NXT*RHO_NXT * 0 + Rhoid_y + RHO_NXT * Rhoid_z];} //[k+2][j+2][im+2];}
     else bc_xm = 0.0;
     if (Rhoid_y==0)         {bc_ym = reinterpret_cast<real_type*>(shared_mem)[RHO_NXT*RHO_NXT * 1 + Rhoid_x + RHO_NXT * Rhoid_z];} //[k+2][j+2][im+2];}
     else bc_ym = 0.0;
     if (Rhoid_z==0)         {bc_zm = reinterpret_cast<real_type*>(shared_mem)[RHO_NXT*RHO_NXT * 2 + Rhoid_x + RHO_NXT * Rhoid_y];} //[k+2][j+2][im+2];}
     else bc_zm = 0.0;
	 
     if (Rhoid_x==RHO_NXT-1) {bc_xp = reinterpret_cast<real_type*>(shared_mem)[RHO_NXT*RHO_NXT * 3 + Rhoid_y + RHO_NXT * Rhoid_z];} //[k+2][j+2][ip+2];}
     else bc_xp = 0.0;
     if (Rhoid_y==RHO_NXT-1) {bc_yp = reinterpret_cast<real_type*>(shared_mem)[RHO_NXT*RHO_NXT * 4 + Rhoid_x + RHO_NXT * Rhoid_z];} //[k+2][j+2][ip+2];}
     else bc_yp = 0.0;
     if (Rhoid_z==RHO_NXT-1) {bc_zp = reinterpret_cast<real_type*>(shared_mem)[RHO_NXT*RHO_NXT * 5 + Rhoid_x + RHO_NXT * Rhoid_y];} //[k+2][j+2][ip+2];}
     else bc_zp = 0.0;

     //     g_Rho_Array[bid][t] = t; 
     g_Rho_Array[bid][t] *= -Const; 
     g_Rho_Array[bid][t] += bc_xm + bc_ym + bc_zm + bc_xp + bc_yp + bc_zp;

     }
     t += stride;    

   } while (t < RHO_NXT*RHO_NXT*RHO_NXT);
   __syncthreads();

  // if (threadIdx.x==0 && threadIdx.y==0 && bid==0){
  //   for (int i = 0; i<RHO_NXT; i++){
  //    for (int j = 0; j<RHO_NXT; j++){
  //      for (int k = 0; k<RHO_NXT; k++){
  // 	 printf("After interp  ijk=%d %d %d ids=%d shared=%.9e \n", i,j,k,(i*RHO_NXT + j)*N + k, g_Rho_Array[bid][(i*RHO_NXT + j)*N + k]);	 
  //  }}}
  // }
  //    __syncthreads();
   

   //  ----------------------------------------------   Discret Sine Transform Poisson Solver -- START -----------------------------------------------//

   //  Forward FFT
   DST_Scheme(ID, Nslab, g_Rho_Array[bid],workspace,0);
   DST_Scheme(ID, Nslab, g_Rho_Array[bid],workspace,1);
   DST_Scheme(ID, Nslab, g_Rho_Array[bid],workspace,2);

  // if (threadIdx.x==0 && threadIdx.y==0 && bid==0){
  //   for (int i = 0; i<RHO_NXT; i++){
  //    for (int j = 0; j<RHO_NXT; j++){
  //      for (int k = 0; k<RHO_NXT; k++){
  // 	 printf("After interp  ijk=%d %d %d ids=%d shared=%.9e \n", i,j,k,(i*RHO_NXT + j)*N + k, g_Rho_Array[bid][(i*RHO_NXT + j)*N + k]);	 
  //  }}}
  // }
  //    __syncthreads();

    // Poisson Eigen
  __shared__ real Eigen[RHO_NXT];
  for (int i=0; i<RHO_NXT; i++)
    Eigen[i] = 1.-COS(M_PI*(i+1)/(RHO_NXT+1));

   __syncthreads();
   
   t=ID;
   do
     {
     Rhoid_z = t /(RHO_NXT*RHO_NXT);
     Rhoid_y = (t /RHO_NXT) % RHO_NXT;
     Rhoid_x = t % RHO_NXT;
     g_Rho_Array[bid][t] /=  2. * (Eigen[Rhoid_x] + Eigen[Rhoid_y] + Eigen[Rhoid_z]) * (8*(N+1)*(N+1)*(N+1));

     t +=stride;

     } while (t < RHO_NXT*RHO_NXT*RHO_NXT);
     __syncthreads();
 
   // // reverse FFT  
   DST_Scheme(ID, Nslab, g_Rho_Array[bid],workspace,0);
   DST_Scheme(ID, Nslab, g_Rho_Array[bid],workspace,1);
   DST_Scheme(ID, Nslab, g_Rho_Array[bid],workspace,2);
    
  // if (threadIdx.x==0 && threadIdx.y==0 && bid==0){
  //   for (int i = 0; i<RHO_NXT; i++){
  //    for (int j = 0; j<RHO_NXT; j++){
  //      for (int k = 0; k<RHO_NXT; k++){
  // 	 printf("After interp  ijk=%d %d %d ids=%d shared=%.9e \n", i,j,k,(i*RHO_NXT + j)*N + k, g_Rho_Array[bid][(i*RHO_NXT + j)*N + k] / (8*(N+1)*(N+1)*(N+1)));	 
  //  }}}
  // }
  //    __syncthreads();

   //  ----------------------------------------------   Discret Sine Transform Poisson Solver -- END  -----------------------------------------------//

   t=ID;
   do
   {
     Rhoid_z = t /(GRA_NXT*GRA_NXT);
     Rhoid_y = (t /GRA_NXT) % GRA_NXT;
     Rhoid_x = t % GRA_NXT;

     Rhoid    =  (Rhoid_z + 2) * (RHO_NXT) * (RHO_NXT)
               + (Rhoid_y + 2) * (RHO_NXT)
               + (Rhoid_x + 2) ;

     if (t< GRA_NXT * GRA_NXT * GRA_NXT){
       g_Pot_Array_Out[bid][t] = g_Rho_Array[bid][Rhoid] ; /// (8*(N+1)*(N+1)*(N+1));
     }
     t += stride;

   } while (t< GRA_NXT * GRA_NXT * GRA_NXT);
    __syncthreads();
} // FUNCTION : CUPOT_PoissonSolver_SOR



#endif // #if ( defined GRAVITY  &&  defined GPU  &&  POT_SCHEME == SOR )
